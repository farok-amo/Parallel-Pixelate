#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <cstring>

#include "cuda.cuh"
#include "hip/hip_runtime.h"
#include ""

#include "helper.h"

///
/// Algorithm storage
///
// Host copy of input image
Image cuda_input_image;
// Host copy of image tiles in each dimension
unsigned int cuda_TILES_X, cuda_TILES_Y;
// Pointer to device buffer for calculating the sum of each tile mosaic, this must be passed to a kernel to be used on device
unsigned long long* d_mosaic_sum;
// Pointer to device buffer for storing the output pixels of each tile, this must be passed to a kernel to be used on device
unsigned char* d_mosaic_value;
// Pointer to device image data buffer, for storing the input image, this must be passed to a kernel to be used on device
unsigned char* d_input_image_data;
// Pointer to device image data buffer, for storing the output image data, this must be passed to a kernel to be used on device
unsigned char* d_output_image_data;
// Pointer to device buffer for the global pixel average sum, this must be passed to a kernel to be used on device
unsigned long long* d_global_pixel_sum;
// Host output image
Image cuda_output_image;
// Host mosaic sum 
unsigned long long* cuda_mosaic_sum;
// Host mosaic value
unsigned char* cuda_mosaic_value;
// Host global pixel sum
unsigned long long* cuda_global_pixel_sum;
// Device variables
__device__ int d_input_image_channels;
__device__ unsigned int d_TILES_X;
__device__ unsigned int d_TILES_Y;
__device__ unsigned int d_input_image_width; // To use for pixel offset calculating

void cuda_begin(const Image* input_image) {
    // These are suggested CUDA memory allocations that match the CPU implementation
    // If you would prefer, you can rewrite this function (and cuda_end()) to suit your preference
    cuda_TILES_X = input_image->width / TILE_SIZE;
    cuda_TILES_Y = input_image->height / TILE_SIZE;

    // Allocate buffer for calculating the sum of each tile mosaic
    CUDA_CALL(hipMalloc(&d_mosaic_sum, cuda_TILES_X * cuda_TILES_Y * input_image->channels * sizeof(unsigned long long)));

    // Allocate buffer for storing the output pixel value of each tile
    CUDA_CALL(hipMalloc(&d_mosaic_value, cuda_TILES_X * cuda_TILES_Y * input_image->channels * sizeof(unsigned char)));

    const size_t image_data_size = input_image->width * input_image->height * input_image->channels * sizeof(unsigned char);
    // Allocate copy of input image
    cuda_input_image = *input_image;
    cuda_input_image.data = (unsigned char*)malloc(image_data_size);
    memcpy(cuda_input_image.data, input_image->data, image_data_size);

    // Allocate and fill device buffer for storing input image data
    CUDA_CALL(hipMalloc(&d_input_image_data, image_data_size));
    CUDA_CALL(hipMemcpy(d_input_image_data, input_image->data, image_data_size, hipMemcpyHostToDevice));

    // Allocate device buffer for storing output image data
    CUDA_CALL(hipMalloc(&d_output_image_data, image_data_size));

    // Allocate and zero buffer for calculation global pixel average
    CUDA_CALL(hipMalloc(&d_global_pixel_sum, input_image->channels * sizeof(unsigned long long)));

    // Copy host variables to the host
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_input_image_width), &input_image->width, sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_input_image_channels), &input_image->channels, sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_TILES_X), &cuda_TILES_X, sizeof(unsigned int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_TILES_Y), &cuda_TILES_Y, sizeof(unsigned int)));

    // Allocate host mosaic sum
    cuda_mosaic_sum = (unsigned long long*)malloc(cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned long long));
    // Allocate host mosaic value
    cuda_mosaic_value = (unsigned char*)malloc(cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned char));
    // Allocate host global pixel 
    cuda_global_pixel_sum = (unsigned long long*)malloc(cuda_input_image.channels * sizeof(unsigned long long));
    // Allocate host output image 
    cuda_output_image.data = (unsigned char*)malloc(image_data_size);
  
}

__global__ void kernel_stage1(unsigned char* d_input_image_data, unsigned long long* d_mosaic_sum) {

    unsigned int t_x = blockIdx.x;
    unsigned int t_y = blockIdx.y;
    unsigned int p_x = threadIdx.x;
    unsigned int p_y = threadIdx.y;

    const unsigned int tile_index = (t_y * d_TILES_X + t_x) * d_input_image_channels;
    const unsigned int tile_offset = (t_y * d_TILES_X * TILE_SIZE * TILE_SIZE + t_x * TILE_SIZE) * d_input_image_channels;
    const unsigned int pixel_offset = (p_y * d_input_image_width + p_x) * d_input_image_channels;

    unsigned int r_sum = d_input_image_data[tile_offset + pixel_offset];
    unsigned int g_sum = d_input_image_data[tile_offset + pixel_offset + 1];
    unsigned int b_sum = d_input_image_data[tile_offset + pixel_offset + 2];

    for (int offset = 16; offset > 0; offset /= 2) {
        r_sum += __shfl_down(r_sum, offset);
        g_sum += __shfl_down(g_sum, offset);
        b_sum += __shfl_down(b_sum, offset);
    }

    if (threadIdx.x % 32 == 0) {
        //avoiding loop usage since channels are always 3 (R, G, B), improves performence and code coherency  
        atomicAdd(&d_mosaic_sum[tile_index], r_sum);
        atomicAdd(&d_mosaic_sum[tile_index + 1], g_sum);
        atomicAdd(&d_mosaic_sum[tile_index + 2], b_sum);
    }
    //Avoid branch divergance from above
    __syncthreads();
}

void cuda_stage1() {
    // Optionally during development call the skip function with the correct inputs to skip this stage
   // skip_tile_sum(&cuda_input_image, d_mosaic_sum);

    dim3 blocks_per_grid(cuda_TILES_X, cuda_TILES_Y, 1);
    dim3 threads_per_block(TILE_SIZE, TILE_SIZE, 1);

    kernel_stage1 <<<blocks_per_grid, threads_per_block >>>(d_input_image_data, d_mosaic_sum);

    // No Need to retrieve mosaic_sum since we can use d_mosaic_sum

#ifdef VALIDATION
    // TODO: Uncomment and call the validation function with the correct inputs
     // You will need to copy the data back to host before passing to these functions
     // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    // validate_tile_sum(&cuda_input_image, cuda_mosaic_sum);
#endif
}

__global__ void kernel_stage2(unsigned long long* d_mosaic_sum, unsigned char* d_mosaic_value, unsigned long long* d_global_pixel_sum) {
    unsigned int t = blockIdx.x * blockDim.x + threadIdx.x;

    d_mosaic_value[t * d_input_image_channels] = (unsigned char)(d_mosaic_sum[t * d_input_image_channels] / TILE_PIXELS);
    d_mosaic_value[t * d_input_image_channels + 1] = (unsigned char)(d_mosaic_sum[t * d_input_image_channels + 1] / TILE_PIXELS);
    d_mosaic_value[t * d_input_image_channels + 2] = (unsigned char)(d_mosaic_sum[t * d_input_image_channels + 2] / TILE_PIXELS);

    unsigned int r_sum = d_mosaic_value[t * d_input_image_channels];
    unsigned int g_sum = d_mosaic_value[t * d_input_image_channels + 1];
    unsigned int b_sum = d_mosaic_value[t * d_input_image_channels + 2];

    for (int offset = 16; offset > 0; offset /= 2) {
        r_sum += __shfl_down(r_sum, offset);
        g_sum += __shfl_down(g_sum, offset);
        b_sum += __shfl_down(b_sum, offset);
    }

    if (threadIdx.x % 32 == 0) {
        atomicAdd(&d_global_pixel_sum[0], r_sum);
        atomicAdd(&d_global_pixel_sum[1], g_sum);
        atomicAdd(&d_global_pixel_sum[2], b_sum);
    }
}
void cuda_stage2(unsigned char* output_global_average) {
    // Optionally during development call the skip function with the correct inputs to skip this stage
    // skip_compact_mosaic(cuda_TILES_X, cuda_TILES_Y, d_mosaic_sum, d_mosaic_value, output_global_average);

    unsigned int total_tiles = cuda_TILES_X * cuda_TILES_Y;
    unsigned int grid_size = (unsigned int)ceil(((double)(total_tiles)) / 1024);
    unsigned int threads_n = (unsigned int)ceil((double)total_tiles / grid_size);

    dim3 blocks_per_grid(grid_size);
    dim3 threads_per_block(threads_n);

    kernel_stage2<<<blocks_per_grid, threads_per_block >>>(d_mosaic_sum, d_mosaic_value, d_global_pixel_sum);

    CUDA_CALL(hipMemcpy(cuda_global_pixel_sum, d_global_pixel_sum, cuda_input_image.channels * sizeof(unsigned long long), hipMemcpyDeviceToHost));

    //Recombine into main host variable
    output_global_average[0] = (unsigned char)(cuda_global_pixel_sum[0] / (total_tiles));
    output_global_average[1] = (unsigned char)(cuda_global_pixel_sum[1] / (total_tiles));
    output_global_average[2] = (unsigned char)(cuda_global_pixel_sum[2] / (total_tiles));

#ifdef VALIDATION
    // TODO: Uncomment and call the validation function with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    // validate_compact_mosaic(cuda_TILES_X, cuda_TILES_Y, cuda_mosaic_sum, cuda_mosaic_value, output_global_average);
#endif    
}

__global__ void kernel_stage3(unsigned char* d_output_image_data, unsigned char* d_mosaic_value) {

    unsigned int t_x = blockIdx.x;
    unsigned int t_y = blockIdx.y;
    unsigned int p_x = threadIdx.x;
    unsigned int p_y = threadIdx.y;

    const unsigned int tile_index = (t_y * d_TILES_X + t_x) * d_input_image_channels;
    const unsigned int tile_offset = (t_y * d_TILES_X * TILE_SIZE * TILE_SIZE + t_x * TILE_SIZE) * d_input_image_channels;
    const unsigned int pixel_offset = (p_y * d_input_image_width + p_x) * d_input_image_channels;

    d_output_image_data[tile_offset + pixel_offset] = d_mosaic_value[tile_index];
    d_output_image_data[tile_offset + pixel_offset + 1] = d_mosaic_value[tile_index + 1];
    d_output_image_data[tile_offset + pixel_offset + 2] = d_mosaic_value[tile_index + 2];
}
void cuda_stage3() {
    // Broadcast the compact mosaic pixels back out to the full image size
    // Optionally during development call the skip function with the correct inputs to skip this stage
    // skip_broadcast(&cuda_input_image, d_mosaic_value, &cuda_input_image);

    dim3 blocks_per_grid(cuda_TILES_X, cuda_TILES_Y, 1);
    dim3 threads_per_block(TILE_SIZE, TILE_SIZE, 1);

    kernel_stage3<<<blocks_per_grid, threads_per_block >>>(d_output_image_data, d_mosaic_value);

#ifdef VALIDATION
    // TODO: Uncomment and call the validation function with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    // validate_broadcast(&cuda_input_image, cuda_mosaic_value, &cuda_output_image);
#endif    
}
void cuda_end(Image* output_image) {
    // This function matches the provided cuda_begin(), you may change it if desired

    // Store return value
    output_image->width = cuda_input_image.width;
    output_image->height = cuda_input_image.height;
    output_image->channels = cuda_input_image.channels;
    CUDA_CALL(hipMemcpy(output_image->data, d_output_image_data, output_image->width * output_image->height * output_image->channels * sizeof(unsigned char), hipMemcpyDeviceToHost));
    // Release allocations
    free(cuda_input_image.data);
    CUDA_CALL(hipFree(d_mosaic_value));
    CUDA_CALL(hipFree(d_mosaic_sum));
    CUDA_CALL(hipFree(d_input_image_data));
    CUDA_CALL(hipFree(d_output_image_data));
    CUDA_CALL(hipFree(d_global_pixel_sum));
    free(cuda_mosaic_sum);
    free(cuda_mosaic_value);
    free(cuda_output_image.data);
    free(cuda_global_pixel_sum);
}
